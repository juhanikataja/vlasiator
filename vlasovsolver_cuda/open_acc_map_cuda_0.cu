#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_header.cuh"
#include "open_acc_map_h.cuh"
#include "../vlasovsolver/vec.h"
#include "../definitions.h"

//#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NPP_MAXABS_32F ( 3.402823466e+38f )
#define NPP_MINABS_32F ( 1.175494351e-38f )
#define NPP_MAXABS_64F ( 1.7976931348623158e+308 )
#define NPP_MINABS_64F ( 2.2250738585072014e-308 )

#define i_pcolumnv_cuda(j, k, k_block, num_k_blocks) ( ((j) / ( VECL / WID)) * WID * ( num_k_blocks + 2) + (k) + ( k_block + 1 ) * WID )
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ));
static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
__device__ Vec minmod(const Vec slope1, const Vec slope2)
{
  const Vec zero(0.0);
  Vec slope = select(abs(slope1) < abs(slope2), slope1, slope2);
  return select(slope1 * slope2 <= 0, zero, slope);
}
__device__ Vec maxmod(const Vec slope1, const Vec slope2)
{
  const Vec zero(0.0);
  Vec slope = select(abs(slope1) > abs(slope2), slope1, slope2);
  return select(slope1 * slope2 <= 0, zero, slope);
}
__device__ Vec slope_limiter_sb(const Vec &l, const Vec &m, const Vec &r)
{
  Vec a = r-m;
  Vec b = m-l;
  const Vec slope1 = minmod(a, 2*b);
  const Vec slope2 = minmod(2*a, b);
  return maxmod(slope1, slope2);
}
__device__ Vec slope_limiter(const Vec &l, const Vec &m,const Vec &r)
{
   return slope_limiter_sb(l,m,r);
}
__device__ void compute_plm_coeff(const Vec * const values, uint k, Vec a[2], const Realv threshold)
{
  // scale values closer to 1 for more accurate slope limiter calculation
  const Realv scale = 1./threshold;
  //Vec v_1 = values[k - 1] * scale;
  //Vec v_2 = values[k] * scale;
  //Vec v_3 = values[k + 1] * scale;
  //Vec d_cv = slope_limiter(v_1, v_2, v_3) * threshold;
  const Vec d_cv = slope_limiter( values[k-1]*scale, values[k]*scale, values[k+1]*scale)*threshold;
  a[0] = values[k] - d_cv * 0.5;
  a[1] = d_cv * 0.5;
}

__global__ void acceleration_1
(
  double *dev_blockData,
  Column *dev_columns,
  Vec *dev_values,
  int *dev_cell_indices_to_id,
  int totalColumns,
  Realv intersection,
  Realv intersection_di,
  Realv intersection_dj,
  Realv intersection_dk,
  Realv v_min,
  Realv i_dv,
  Realv dv,
  Realv minValue,
  int acc_semilag_flag,
  int bdsw3
)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if(index == 0)
  {
    //printf("CUDA 1 Kernel\n");
    for( uint column=0; column < totalColumns; column++)
    {
      //printf("CUDA 2\n");
      // i,j,k are relative to the order in which we copied data to the values array.
      // After this point in the k,j,i loops there should be no branches based on dimensions
      // Note that the i dimension is vectorized, and thus there are no loops over i
      // Iterate through the perpendicular directions of the column
       for (uint j = 0; j < WID; j += VECL/WID)
       {
         //printf("CUDA 3; VECL = %d\n", VECL);
          const vmesh::LocalID nblocks = dev_columns[column].nblocks;
          // create vectors with the i and j indices in the vector position on the plane.
          #if VECL == 4
            const Veci i_indices = Veci(0, 1, 2, 3);
            const Veci j_indices = Veci(j, j, j, j);
          #elif VECL == 8
            const Veci i_indices = Veci(0, 1, 2, 3, 0, 1, 2, 3);
            const Veci j_indices = Veci(j, j, j, j, j + 1, j + 1, j + 1, j + 1);
          #elif VECL == 16
            const Veci i_indices = Veci(0, 1, 2, 3,
                                        0, 1, 2, 3,
                                        0, 1, 2, 3,
                                        0, 1, 2, 3);
            const Veci j_indices = Veci(j, j, j, j,
                                        j + 1, j + 1, j + 1, j + 1,
                                        j + 2, j + 2, j + 2, j + 2,
                                        j + 3, j + 3, j + 3, j + 3);
          #endif

          const Veci  target_cell_index_common =
             i_indices * dev_cell_indices_to_id[0] +
             j_indices * dev_cell_indices_to_id[1];

          // intersection_min is the intersection z coordinate (z after
          // swaps that is) of the lowest possible z plane for each i,j
          // index (i in vector)
          const Vec intersection_min =
             intersection +
             (dev_columns[column].i * WID + to_realv(i_indices)) * intersection_di +
             (dev_columns[column].j * WID + to_realv(j_indices)) * intersection_dj;

          /*compute some initial values, that are used to set up the
           * shifting of values as we go through all blocks in
           * order. See comments where they are shifted for
           * explanations of their meaning*/
          Vec v_r0( (WID * dev_columns[column].kBegin) * dv + v_min);
          Vec lagrangian_v_r0((v_r0-intersection_min)/intersection_dk);

          /* compute location of min and max, this does not change for one
          column (or even for this set of intersections, and can be used
          to quickly compute max and min later on*/
          //TODO, these can be computed much earlier, since they are
          //identiacal for each set of intersections

          int minGkIndex=0, maxGkIndex=0; // 0 for compiler
          {
            Realv maxV = (sizeof(Realv) == 4) ? NPP_MINABS_32F : NPP_MINABS_64F;
            Realv minV = (sizeof(Realv) == 4) ? NPP_MAXABS_32F : NPP_MAXABS_64F;
             for(int i = 0; i < VECL; i++)
             {
                if (lagrangian_v_r0[i] > maxV)
                {
                   maxV = lagrangian_v_r0[i];
                   maxGkIndex = i;
                }
                if (lagrangian_v_r0[i] < minV)
                {
                   minV = lagrangian_v_r0[i];
                   minGkIndex = i;
                }
             }
          }
          // loop through all blocks in column and compute the mapping as integrals.
          for (uint k=0; k < WID * nblocks; ++k)
          {
             //printf("CUDA 4\n");
             // Compute reconstructions
             // values + i_pcolumnv(n_cblocks, -1, j, 0) is the starting point of the column data for fixed j
             // k + WID is the index where we have stored k index, WID amount of padding.
             //if(acc_semilag_flag==0)
             //{
              Vec a[2];
              compute_plm_coeff(dev_values + dev_columns[column].valuesOffset + i_pcolumnv_cuda(j, 0, -1, nblocks), k + WID, a, minValue);
              /*
              if(column==0 && j==0 && k==0)
              for(int a_1 = 0; a_1 < VECL; a_1++)
              {
                  printf("CUDA a[0]: column = %d; j = %d; k = %d; a[%d] = %.12e\n", column, j, k, a_1, a[0][a_1]);
                  printf("CUDA a[1]: column = %d; j = %d; k = %d; a[%d] = %.12e\n", column, j, k, a_1, a[1][a_1]);
              }
              */
              //}
              /*
              if(acc_semilag_flag==1)
              {
                Vec *a = new Vec[3];
                //compute_ppm_coeff(values + dev_columns[column].valuesOffset  + i_pcolumnv(j, 0, -1, nblocks), h4, k + WID, a, minValue);
              }
              if(acc_semilag_flag==2)
              {
                Vec *a = new Vec[5];
                //compute_pqm_coeff(values + dev_columns[column].valuesOffset  + i_pcolumnv(j, 0, -1, nblocks), h8, k + WID, a, minValue);
              }
              */
             // set the initial value for the integrand at the boundary at v = 0
             // (in reduced cell units), this will be shifted to target_density_1, see below.
             Vec target_density_r(0.0);
             /*
             if(column==0 && j==0 && k==0)
             {
               for(int a_1 = 0; a_1 < VECL; a_1++)
               {
                 printf("CUDA 0: target_density_r [%d] = %.2f\n", a_1, target_density_r[a_1]);
               }
             }
             */
             // v_l, v_r are the left and right velocity coordinates of source cell.
             Vec v_r = v_r0  + (k+1)* dv;
             Vec v_l = v_r0  + k* dv;
             // left(l) and right(r) k values (global index) in the target
             // Lagrangian grid, the intersecting cells. Again old right is new left.

             // I keep only this version with Fallback, because the version with Agner requires another call to CPU
             Veci lagrangian_gk_l = truncate_to_int((v_l-intersection_min)/intersection_dk);
             Veci lagrangian_gk_r = truncate_to_int((v_r-intersection_min)/intersection_dk);
             //limits in lagrangian k for target column. Also take into
             //account limits of target column
             int minGk = max(int(lagrangian_gk_l[minGkIndex]), int(dev_columns[column].minBlockK * WID));
             int maxGk = min(int(lagrangian_gk_r[maxGkIndex]), int((dev_columns[column].maxBlockK + 1) * WID - 1));
             // Run along the column and perform the polynomial reconstruction
             for(int gk = dev_columns[column].minBlockK * WID; gk <= dev_columns[column].maxBlockK * WID; gk++)
             {
                /*
                if(column==0 && j==0 && k==0)
                {
                  printf("CUDA 1: minGk = %d; gk = %d; maxGk = %d;\n", minGk, gk, maxGk);
                }
                */
                if(gk < minGk || gk > maxGk)
                { continue; }
                //printf("CUDA 6\n");
                const int blockK = gk/WID;
                const int gk_mod_WID = (gk - blockK * WID);
                /*
                if(column==0 && j==0 && k==0)
                {
                 printf("CUDA 2: blockK = %d; gk_mod_WID = %d;\n", blockK, gk_mod_WID);
                }
                */
                //the block of the Lagrangian cell to which we map
                //const int target_block(target_block_index_common + blockK * block_indices_to_id[2]);
                //cell indices in the target block  (TODO: to be replaced by
                //compile time generated scatter write operation)
                const Veci target_cell(target_cell_index_common + gk_mod_WID * dev_cell_indices_to_id[2]);
                /*
                if(column==0 && j==0 && k==0)
                {
                  for(int a_1 = 0; a_1 < VECL; a_1++)
                  {
                    printf("CUDA 3: target_cell [%d] = %d\n", a_1, target_cell[a_1]);
                  }
                }
                */
                //the velocity between which we will integrate to put mass
                //in the targe cell. If both v_r and v_l are in same cell
                //then v_1,v_2 should be between v_l and v_r.
                //v_1 and v_2 normalized to be between 0 and 1 in the cell.
                //For vector elements where gk is already larger than needed (lagrangian_gk_r), v_2=v_1=v_r and thus the value is zero.
                const Vec v_norm_r = (  min(  max( (gk + 1) * intersection_dk + intersection_min, v_l), v_r) - v_l) * i_dv;
                /*
                if(column==0 && j==0 && k==0)
                {
                  printf("CUDA CHECK 1: gk = %d; intersection_dk = %d; intersection_min = %d; i_dv = %d;\n", gk, intersection_dk, intersection_min, i_dv);
                  for(int a_1 = 0; a_1 < VECL; a_1++)
                  {
                    printf("CUDA CHECK 2: v_l [%d] = %.2f\n", a_1, v_l[a_1]);
                    printf("CUDA CHECK 3: v_r [%d] = %.2f\n", a_1, v_r[a_1]);
                  }
                }
                */
                /*
                if(column==0 && j==0 && k==0)
                {
                  for(int a_1 = 0; a_1 < VECL; a_1++)
                  {
                    printf("CUDA 4: v_norm_r [%d] = %.2f\n", a_1, v_norm_r[a_1]);
                  }
                }
                */
                /*shift, old right is new left*/
                const Vec target_density_l = target_density_r;
                /*
                if(column==0 && j==0 && k==0)
                {
                  for(int a_1 = 0; a_1 < VECL; a_1++)
                  {
                    printf("CUDA 5: target_density_l [%d] = %.2f\n", a_1, target_density_l[a_1]);
                  }
                }
                */
                // compute right integrand
                if(acc_semilag_flag==0)
                  target_density_r = v_norm_r * ( a[0] + v_norm_r * a[1] );
                //if(acc_semilag_flag==1)
                //  target_density_r = v_norm_r * ( a[0] + v_norm_r * ( a[1] + v_norm_r * a[2] ) );
                //if(acc_semilag_flag==2)
                //  target_density_r =
                //    v_norm_r * ( a[0] + v_norm_r * ( a[1] + v_norm_r * ( a[2] + v_norm_r * ( a[3] + v_norm_r * a[4] ) ) ) );
                //store values, one element at a time. All blocks have been created by now.
                //TODO replace by vector version & scatter & gather operation
                const Vec target_density = target_density_r - target_density_l;
                for (int target_i=0; target_i < VECL; ++target_i)
                {
                  //printf("CUDA 8\n");
                  // do the conversion from Realv to Realf here, faster than doing it in accumulation
                  const Realf tval = target_density[target_i];
                  const uint tcell = target_cell[target_i];
                  //if(target_i == ( (VECL) - 1) )
                  //  printf("CUDA: tval = %.2f; tcell = %d;\n", tval, tcell);
                  //printf("&dev_blockData[a] = %.2f; tcell = %d\n", dev_blockData[dev_columns[column].targetBlockOffsets[blockK]], tcell );
                  (&dev_blockData[dev_columns[column].targetBlockOffsets[blockK]])[tcell] += tval;
                  //for(uint cell=0; cell<bdsw3; cell++)
                  //{
                  //  printf("blockData[cell] = %.2f\n", blockData[cell]);
                  //}
                  //for(uint aa=0; aa<bdsw3; aa++)
                  //{
                  //    printf("dev_blockData[%d] = %.2f\n", aa, dev_blockData[aa]);
                  //}
                  //printf("CUDA 11\n");
                }  // for-loop over vector elements
             } // for loop over target k-indices of current source block
          } // for-loop over source blocks
       } //for loop over j index
    } //for loop over columns
  }
}

Realf* acceleration_1_wrapper
(
  Realf *blockData,
  Column *columns,
  Vec *values,
  uint cell_indices_to_id[],
  int totalColumns,
  int valuesSizeRequired,
  int bdsw3,
  Realv intersection,
  Realv intersection_di,
  Realv intersection_dj,
  Realv intersection_dk,
  Realv v_min,
  Realv i_dv,
  Realv dv,
  Realv minValue
)
{
  int acc_semilag_flag = 0;
  #ifdef ACC_SEMILAG_PLM
    acc_semilag_flag = 0;
  #endif
  #ifdef ACC_SEMILAG_PPM
    acc_semilag_flag = 1;
  #endif
  #ifdef ACC_SEMILAG_PQM
    acc_semilag_flag = 2;
  #endif

  double *dev_blockData;
  HANDLE_ERROR( hipMalloc((void**)&dev_blockData, bdsw3*sizeof(double)) );
  HANDLE_ERROR( hipMemcpy(dev_blockData, blockData, bdsw3*sizeof(double), hipMemcpyHostToDevice) );

  Column *dev_columns;
  HANDLE_ERROR( hipMalloc((void**)&dev_columns, totalColumns*sizeof(Column)) );
  HANDLE_ERROR( hipMemcpy(dev_columns, columns, totalColumns*sizeof(Column), hipMemcpyHostToDevice) );

  int *dev_cell_indices_to_id;
  HANDLE_ERROR( hipMalloc((void**)&dev_cell_indices_to_id, 3*sizeof(int)) );
  HANDLE_ERROR( hipMemcpy(dev_cell_indices_to_id, cell_indices_to_id, 3*sizeof(int), hipMemcpyHostToDevice) );

  Vec *dev_values;
  HANDLE_ERROR( hipMalloc((void**)&dev_values, valuesSizeRequired*sizeof(Vec)) );
  HANDLE_ERROR( hipMemcpy(dev_values, values, valuesSizeRequired*sizeof(Vec), hipMemcpyHostToDevice) );

  hipLaunchKernelGGL(acceleration_1, BLOCKS, THREADS, 0, 0, 
    dev_blockData,
    dev_columns,
    dev_values,
    dev_cell_indices_to_id,
        totalColumns,
        intersection,
        intersection_di,
        intersection_dj,
        intersection_dk,
        v_min,
        i_dv,
        dv,
        minValue,
        acc_semilag_flag,
        bdsw3
  );

  hipDeviceSynchronize();
  HANDLE_ERROR( hipMemcpy(blockData, dev_blockData, bdsw3*sizeof(double), hipMemcpyDeviceToHost) );

  HANDLE_ERROR( hipFree(dev_blockData) );
  HANDLE_ERROR( hipFree(dev_cell_indices_to_id) );
  HANDLE_ERROR( hipFree(dev_columns) );
  HANDLE_ERROR( hipFree(dev_values) );

  return blockData;
}
